#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>

#include "src/flashinfer_ops.cuh"

#include "capi.h"

__attribute__ ((visibility("default")))
void flashinfer_BatchDecodeHandlerPlan(
    flashinfer_BatchDecodeHandler* handler,
    void* float_buffer,
    size_t float_workspace_size_in_bytes,
    void* int_buffer,
    size_t int_workspace_size_in_bytes,
    int32_t* indptr_h,
    int32_t* last_page_len_h,
    uint32_t batch_size,
    uint32_t num_qo_heads,
    uint32_t num_kv_heads,
    uint32_t head_dim,
    uint32_t page_size
) {
    using DTypeQ = __hip_bfloat16;
    using DTypeKV = DTypeQ;
    using DTypeO = DTypeQ;
    using IdType = int32_t;

    static_assert(sizeof(flashinfer::BatchDecodeHandler) == sizeof(flashinfer_BatchDecodeHandler), "mismatch");

    flashinfer::BatchDecodeHandlerPlan<DTypeQ, DTypeKV, DTypeO, IdType>(
        reinterpret_cast<flashinfer::BatchDecodeHandler*>(handler),
        float_buffer,
        float_workspace_size_in_bytes,
        int_buffer,
        int_workspace_size_in_bytes,
        indptr_h,
        last_page_len_h,
        batch_size,
        num_qo_heads,
        num_kv_heads,
        head_dim,
        page_size,
        flashinfer::PosEncodingMode::kRoPELlama);
}

__attribute__ ((visibility("default")))
void flashinfer_BatchDecodeWithPagedKVCacheWrapper(
    flashinfer_BatchDecodeHandler* handler,
    void* q,
    int32_t* q_rope_offset,
    flashinfer_paged_kv_t paged_kv,
    void* o,
    float* lse,
    uint32_t num_qo_heads,
    float sm_scale,
    float rope_scale,
    float rope_theta,
    void* stream
) {
    using DTypeQ = __hip_bfloat16;
    using DTypeKV = DTypeQ;
    using DTypeO = DTypeQ;
    using IdType = int32_t;

    // hipError_t status =
    flashinfer::BatchDecodeWithPagedKVCacheWrapper<DTypeQ, DTypeKV, DTypeO, IdType>(
        reinterpret_cast<flashinfer::BatchDecodeHandler*>(handler),
        static_cast<DTypeQ*>(q),
        q_rope_offset,
        flashinfer::paged_kv_t<DTypeKV, IdType>(
            paged_kv.num_heads,
            paged_kv.page_size,
            paged_kv.head_dim,
            paged_kv.batch_size,
            flashinfer::QKVLayout::kNHD,
            static_cast<DTypeKV*>(paged_kv.k_data),
            static_cast<DTypeKV*>(paged_kv.v_data),
            paged_kv.kv_strides,
            paged_kv.indices,
            paged_kv.indptr,
            paged_kv.last_page_len,
            paged_kv.rope_pos_offset
        ),
        static_cast<DTypeO*>(o),
        lse,
        num_qo_heads,
        flashinfer::PosEncodingMode::kRoPELlama);
}

__attribute__ ((visibility("default")))
void flashinfer_BatchPrefillHandlerPlan(
    flashinfer_BatchPrefillHandler* handler,
    void* float_buffer,
    size_t float_workspace_size_in_bytes,
    void* int_buffer,
    size_t int_workspace_size_in_bytes,
    int32_t* qo_indptr_h,
    int32_t* kv_indptr_h,
    uint32_t total_num_rows,
    uint32_t batch_size,
    uint32_t num_qo_heads,
    uint32_t num_kv_heads,
    uint32_t head_dim,
    uint32_t page_size
) {
    using DTypeQ = __hip_bfloat16;
    using DTypeKV = DTypeQ;
    using DTypeO = DTypeQ;
    using IdType = int32_t;

    static_assert(sizeof(flashinfer::BatchPrefillHandler) == sizeof(flashinfer_BatchPrefillHandler), "mismatch");

    reinterpret_cast<flashinfer::BatchPrefillHandler*>(handler)->Plan<DTypeQ, IdType>(
        float_buffer,
        float_workspace_size_in_bytes,
        int_buffer,
        int_workspace_size_in_bytes,
        qo_indptr_h,
        kv_indptr_h,
        total_num_rows,
        batch_size,
        num_qo_heads,
        num_kv_heads,
        head_dim,
        page_size);
}

__attribute__ ((visibility("default")))
void flashinfer_BatchPrefillWithPagedKVCacheWrapper(
    flashinfer_BatchPrefillHandler* handler,
    void* q,
    int32_t* qo_indptr,
    int32_t* q_rope_offset,
    flashinfer_paged_kv_t paged_kv,
    void* o,
    float* lse,
    uint32_t num_qo_heads,
    bool causal,
    float sm_scale,
    float rope_scale,
    float rope_theta,
    void* stream
) {
    using DTypeQ = __hip_bfloat16;
    using DTypeKV = DTypeQ;
    using DTypeO = DTypeQ;
    using IdType = int32_t;

    static_assert(sizeof(flashinfer::BatchPrefillHandler) == sizeof(flashinfer_BatchPrefillHandler), "mismatch");

    flashinfer::BatchPrefillWithPagedKVCacheWrapper<DTypeQ, DTypeKV, DTypeO, IdType>(
        reinterpret_cast<flashinfer::BatchPrefillHandler*>(handler),
        static_cast<DTypeQ*>(q),
        qo_indptr,
        q_rope_offset,
        flashinfer::paged_kv_t<DTypeKV, IdType>(
            paged_kv.num_heads,
            paged_kv.page_size,
            paged_kv.head_dim,
            paged_kv.batch_size,
            flashinfer::QKVLayout::kNHD,
            static_cast<DTypeKV*>(paged_kv.k_data),
            static_cast<DTypeKV*>(paged_kv.v_data),
            paged_kv.kv_strides,
            paged_kv.indices,
            paged_kv.indptr,
            paged_kv.last_page_len,
            paged_kv.rope_pos_offset
        ),
        static_cast<DTypeO*>(o),
        lse,
        num_qo_heads,
        causal,
        flashinfer::PosEncodingMode::kRoPELlama,
        false);

}
