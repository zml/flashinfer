#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <flashinfer/cutlass_utils.cuh>

#include "src/flashinfer_ops_sm90.cuh"

#include "capi.h"

__attribute__ ((visibility("default")))
void flashinfer_BatchPrefillHandlerSm90Plan(
    flashinfer_BatchPrefillSm90Handler* handler,
    void* float_buffer,
    size_t float_workspace_size_in_bytes,
    void* int_buffer,
    size_t int_workspace_size_in_bytes,
    int32_t* qo_indptr_h,
    int32_t* kv_indptr_h,
    int32_t* kv_len_arr_h,
    uint32_t total_num_rows,
    uint32_t batch_size,
    uint32_t num_qo_heads,
    uint32_t num_kv_heads,
    uint32_t head_dim,
    uint32_t page_size
) {
    using DTypeQ = flashinfer::cutlass_dtype_t<__hip_bfloat16>;
    using IdType = int32_t;

    static_assert(sizeof(flashinfer::BatchPrefillSm90Handler) == sizeof(flashinfer_BatchPrefillSm90Handler), "mismatch");

    reinterpret_cast<flashinfer::BatchPrefillSm90Handler*>(handler)->Plan<DTypeQ, IdType>(
        float_buffer,
        float_workspace_size_in_bytes,
        int_buffer,
        int_workspace_size_in_bytes,
        qo_indptr_h,
        kv_indptr_h,
        kv_len_arr_h,
        total_num_rows,
        batch_size,
        num_qo_heads,
        num_kv_heads,
        head_dim,
        page_size);
}

__attribute__ ((visibility("default")))
int flashinfer_BatchPrefillWithPagedKVCacheSm90Wrapper(
    flashinfer_BatchPrefillSm90Handler* handler,
    void* q,
    flashinfer_paged_kv_t paged_kv,
    void* o,
    float* lse,
    uint32_t nnz_qo,
    uint32_t num_qo_heads,
    bool causal,
    float sm_scale,
    float rope_scale,
    float rope_theta,
    void* stream
) {
    using DTypeQ = flashinfer::cutlass_dtype_t<__hip_bfloat16>;
    using DTypeKV = DTypeQ;
    using DTypeO = DTypeQ;
    using IdType = int32_t;

    static_assert(sizeof(flashinfer::BatchPrefillSm90Handler) == sizeof(flashinfer_BatchPrefillSm90Handler), "mismatch");

    hipError_t status = flashinfer::BatchPrefillWithPagedKVCacheSm90Wrapper<DTypeQ, DTypeKV, DTypeO, IdType>(
        reinterpret_cast<flashinfer::BatchPrefillSm90Handler*>(handler),
        static_cast<DTypeQ*>(q),
        flashinfer::paged_kv_t<DTypeKV, IdType>(
            paged_kv.num_heads,
            paged_kv.page_size,
            paged_kv.head_dim,
            paged_kv.batch_size,
            flashinfer::QKVLayout::kNHD,
            static_cast<DTypeKV*>(paged_kv.k_data),
            static_cast<DTypeKV*>(paged_kv.v_data),
            paged_kv.kv_strides,
            paged_kv.indices,
            paged_kv.indptr,
            paged_kv.last_page_len,
            paged_kv.rope_pos_offset
        ),
        static_cast<DTypeO*>(o),
        lse,
        nnz_qo,
        num_qo_heads,
        causal,
        flashinfer::PosEncodingMode::kNone,
        std::nullopt,
        1.f,
        1e4,
        reinterpret_cast<hipStream_t>(stream));
    
    return (int)status;
}
