#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = SingleDecodeParams<half, half, half>;

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp,
    hipStream_t stream);

}
    