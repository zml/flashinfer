#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = SinglePrefillParams<half, half, half>;

template hipError_t SinglePrefillWithKVCacheDispatched<64, 64, PosEncodingMode::kNone, 0, MaskMode::kNone, DefaultAttention<
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<64, 64, PosEncodingMode::kNone, 0, MaskMode::kNone, DefaultAttention<
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<64, 64, PosEncodingMode::kNone, 0, MaskMode::kNone, DefaultAttention<
    false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<64, 64, PosEncodingMode::kNone, 0, MaskMode::kNone, DefaultAttention<
    false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp,
    hipStream_t stream);

}
    