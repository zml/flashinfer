#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = BatchPrefillPagedParams<half, half, half, int32_t>;

using AttentionVariant1 = DefaultAttention<false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kNone, AttentionVariant1, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kNone, AttentionVariant1, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kNone, AttentionVariant1, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    

using AttentionVariant2 = DefaultAttention<false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kNone, AttentionVariant2, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kNone, AttentionVariant2, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kNone, AttentionVariant2, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    

using AttentionVariant3 = DefaultAttention<false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kNone, AttentionVariant3, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kNone, AttentionVariant3, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kNone, AttentionVariant3, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    

using AttentionVariant4 = DefaultAttention<false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kNone, AttentionVariant4, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kNone, AttentionVariant4, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kNone, AttentionVariant4, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    

}