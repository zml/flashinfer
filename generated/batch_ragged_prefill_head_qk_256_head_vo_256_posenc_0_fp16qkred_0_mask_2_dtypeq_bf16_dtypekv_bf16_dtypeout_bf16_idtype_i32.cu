#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = BatchPrefillRaggedParams<hip_bfloat16, hip_bfloat16, hip_bfloat16, int32_t>;

using AttentionVariant1 = DefaultAttention<true, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithRaggedKVCacheDispatched<128, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant1, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<64, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant1, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<16, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant1, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
        

using AttentionVariant2 = DefaultAttention<true, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithRaggedKVCacheDispatched<128, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant2, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<64, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant2, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<16, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant2, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
        

using AttentionVariant3 = DefaultAttention<true, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithRaggedKVCacheDispatched<128, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant3, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<64, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant3, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<16, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant3, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
        

using AttentionVariant4 = DefaultAttention<true, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithRaggedKVCacheDispatched<128, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant4, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<64, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant4, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<16, 256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant4, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
        

}
    