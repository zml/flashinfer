#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = BatchPrefillRaggedParams<half, half, half, int32_t>;

using AttentionVariant1 = DefaultAttention<false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithRaggedKVCacheDispatched<128, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant1, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<64, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant1, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<16, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant1, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        

using AttentionVariant2 = DefaultAttention<false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithRaggedKVCacheDispatched<128, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant2, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<64, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant2, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<16, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant2, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        

using AttentionVariant3 = DefaultAttention<false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithRaggedKVCacheDispatched<128, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant3, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<64, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant3, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<16, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant3, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        

using AttentionVariant4 = DefaultAttention<false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithRaggedKVCacheDispatched<128, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant4, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<64, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant4, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        
template hipError_t BatchPrefillWithRaggedKVCacheDispatched<16, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant4, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
        

}
    