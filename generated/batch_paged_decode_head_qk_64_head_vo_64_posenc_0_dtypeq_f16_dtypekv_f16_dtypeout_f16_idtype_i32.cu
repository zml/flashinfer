#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = BatchDecodeParams<half, half, half, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatched<64, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<64, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<64, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<64, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp_v, float* tmp_s,
    hipStream_t stream);

using ParamsMlaT = BatchDecodeParamsMLA<half, half, half, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatchedMLA<64, 8, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, ParamsMlaT>(
    ParamsMlaT params,
    half* tmp_v, float* tmp_s,
    hipStream_t stream);

}
    