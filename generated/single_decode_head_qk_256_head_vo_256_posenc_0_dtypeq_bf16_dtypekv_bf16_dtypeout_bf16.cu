#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = SingleDecodeParams<hip_bfloat16, hip_bfloat16, hip_bfloat16>;

template hipError_t SingleDecodeWithKVCacheDispatched<256, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<256, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<256, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<256, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp,
    hipStream_t stream);

}
    