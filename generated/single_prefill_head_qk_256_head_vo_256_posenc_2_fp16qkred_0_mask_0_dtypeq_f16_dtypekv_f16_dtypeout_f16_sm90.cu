 // single_prefill_sm90 template inst
#include <flashinfer/attention/hopper/default_params.cuh>
#include <flashinfer/attention/hopper/prefill_sm90.cuh>
#include <flashinfer/attention/hopper/variants.cuh>
#include <flashinfer/cutlass_utils.cuh>

namespace flashinfer {

using DTypeQ = cutlass_dtype_t<half>;
using DTypeKV = cutlass_dtype_t<half>;
using DTypeO = cutlass_dtype_t<half>;

using Params = SinglePrefillParams<DTypeQ, DTypeKV, DTypeO>;

template hipError_t SinglePrefillWithKVCacheDispatched
    <256, 256, MaskMode::kNone, /*USE_SLIDING_WINDOW=*/true, LogitsSoftCap, Params>
    (Params& params, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched
    <256, 256, MaskMode::kNone, /*USE_SLIDING_WINDOW=*/false, LogitsSoftCap, Params>
    (Params& params, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched
    <256, 256, MaskMode::kNone, /*USE_SLIDING_WINDOW=*/true, StandardAttention, Params>
    (Params& params, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched
    <256, 256, MaskMode::kNone, /*USE_SLIDING_WINDOW=*/false, StandardAttention, Params>
    (Params& params, hipStream_t stream);

}
    