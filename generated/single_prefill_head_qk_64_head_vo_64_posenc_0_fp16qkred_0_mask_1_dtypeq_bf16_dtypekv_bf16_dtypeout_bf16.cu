#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = SinglePrefillParams<hip_bfloat16, hip_bfloat16, hip_bfloat16>;

template hipError_t SinglePrefillWithKVCacheDispatched<64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, DefaultAttention<
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, DefaultAttention<
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, DefaultAttention<
    false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, DefaultAttention<
    false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp,
    hipStream_t stream);

}
    