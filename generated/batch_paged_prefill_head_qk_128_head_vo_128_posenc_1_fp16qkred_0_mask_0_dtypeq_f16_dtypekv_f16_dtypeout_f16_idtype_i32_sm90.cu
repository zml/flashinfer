 // batch_paged_prefill_sm90 template inst
#include <flashinfer/attention/hopper/default_params.cuh>
#include <flashinfer/attention/hopper/prefill_sm90.cuh>
#include <flashinfer/attention/hopper/variants.cuh>
#include <flashinfer/cutlass_utils.cuh>


namespace flashinfer {

using DTypeQ = cutlass_dtype_t<half>;
using DTypeKV = cutlass_dtype_t<half>;
using DTypeO = cutlass_dtype_t<half>;

using Params = BatchPrefillPagedParams<DTypeQ, DTypeKV, DTypeO, int32_t>;


template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kNone,
     /*USE_SLIDING_WINDOW=*/true,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/true,
     LogitsSoftCap,
     Params>
    (Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kNone,
     /*USE_SLIDING_WINDOW=*/true,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/false,
     LogitsSoftCap,
     Params>
    (Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kNone,
     /*USE_SLIDING_WINDOW=*/false,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/true,
     LogitsSoftCap,
     Params>
    (Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kNone,
     /*USE_SLIDING_WINDOW=*/false,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/false,
     LogitsSoftCap,
     Params>
    (Params& params, hipStream_t stream);
    


template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kNone,
     /*USE_SLIDING_WINDOW=*/true,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/true,
     StandardAttention,
     Params>
    (Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kNone,
     /*USE_SLIDING_WINDOW=*/true,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/false,
     StandardAttention,
     Params>
    (Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kNone,
     /*USE_SLIDING_WINDOW=*/false,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/true,
     StandardAttention,
     Params>
    (Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kNone,
     /*USE_SLIDING_WINDOW=*/false,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/false,
     StandardAttention,
     Params>
    (Params& params, hipStream_t stream);
    

}