#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = SinglePrefillParams<hip_bfloat16, hip_bfloat16, hip_bfloat16>;

template hipError_t SinglePrefillWithKVCacheDispatched<256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, DefaultAttention<
    true, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, DefaultAttention<
    true, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, DefaultAttention<
    true, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<256, 256, PosEncodingMode::kNone, 0, MaskMode::kCustom, DefaultAttention<
    true, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp,
    hipStream_t stream);

}
    