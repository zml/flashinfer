#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = BatchPrefillPagedParams<hip_bfloat16, hip_bfloat16, hip_bfloat16, int32_t>;

using AttentionVariant1 = DefaultAttention<true, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant1, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant1, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant1, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    

using AttentionVariant2 = DefaultAttention<true, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant2, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant2, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant2, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    

using AttentionVariant3 = DefaultAttention<true, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant3, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant3, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant3, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    

using AttentionVariant4 = DefaultAttention<true, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant4, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant4, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCustom, AttentionVariant4, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    

}