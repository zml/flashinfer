#include "hip/hip_runtime.h"
 // single_prefill_sm90 template inst
#include <flashinfer/attention/hopper/default_params.cuh>
#include <flashinfer/attention/hopper/prefill_sm90.cuh>
#include <flashinfer/attention/hopper/variants.cuh>
#include <flashinfer/cutlass_utils.cuh>

namespace flashinfer {

using DTypeQ = cutlass_dtype_t<hip_bfloat16>;
using DTypeKV = cutlass_dtype_t<hip_bfloat16>;
using DTypeO = cutlass_dtype_t<hip_bfloat16>;

using Params = SinglePrefillParams<DTypeQ, DTypeKV, DTypeO>;

template hipError_t SinglePrefillWithKVCacheDispatched
    <64, 64, MaskMode::kCustom, /*USE_SLIDING_WINDOW=*/true, LogitsSoftCap, Params>
    (Params& params, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched
    <64, 64, MaskMode::kCustom, /*USE_SLIDING_WINDOW=*/false, LogitsSoftCap, Params>
    (Params& params, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched
    <64, 64, MaskMode::kCustom, /*USE_SLIDING_WINDOW=*/true, StandardAttention, Params>
    (Params& params, hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched
    <64, 64, MaskMode::kCustom, /*USE_SLIDING_WINDOW=*/false, StandardAttention, Params>
    (Params& params, hipStream_t stream);

}
    