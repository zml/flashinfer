#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = BatchDecodeParams<hip_bfloat16, hip_bfloat16, hip_bfloat16, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    hip_bfloat16* tmp_v, float* tmp_s,
    hipStream_t stream);

using ParamsMlaT = BatchDecodeParamsMLA<hip_bfloat16, hip_bfloat16, hip_bfloat16, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatchedMLA<128, 16, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, ParamsMlaT>(
    ParamsMlaT params,
    hip_bfloat16* tmp_v, float* tmp_s,
    hipStream_t stream);

}
    