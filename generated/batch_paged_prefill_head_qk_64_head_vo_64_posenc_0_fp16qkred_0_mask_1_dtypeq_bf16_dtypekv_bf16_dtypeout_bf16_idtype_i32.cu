#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = BatchPrefillPagedParams<hip_bfloat16, hip_bfloat16, hip_bfloat16, int32_t>;

using AttentionVariant1 = DefaultAttention<false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant1, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant1, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant1, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    

using AttentionVariant2 = DefaultAttention<false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant2, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant2, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant2, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    

using AttentionVariant3 = DefaultAttention<false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant3, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant3, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant3, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    

using AttentionVariant4 = DefaultAttention<false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant4, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant4, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 64, 64, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant4, Params>(
    Params params,
    hip_bfloat16* tmp_v,
    float* tmp_s, hipStream_t stream);
    

}