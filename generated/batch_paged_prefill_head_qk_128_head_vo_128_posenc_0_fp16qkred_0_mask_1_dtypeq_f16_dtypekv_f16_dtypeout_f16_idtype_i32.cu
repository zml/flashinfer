#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = BatchPrefillPagedParams<half, half, half, int32_t>;

using AttentionVariant1 = DefaultAttention<false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant1, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant1, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant1, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    

using AttentionVariant2 = DefaultAttention<false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant2, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant2, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant2, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    

using AttentionVariant3 = DefaultAttention<false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant3, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant3, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant3, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    

using AttentionVariant4 = DefaultAttention<false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>;

template hipError_t BatchPrefillWithPagedKVCacheDispatched<128, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant4, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<64, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant4, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    
template hipError_t BatchPrefillWithPagedKVCacheDispatched<16, 128, 128, PosEncodingMode::kNone, 0, MaskMode::kCausal, AttentionVariant4, Params>(
    Params params,
    half* tmp_v,
    float* tmp_s, hipStream_t stream);
    

}