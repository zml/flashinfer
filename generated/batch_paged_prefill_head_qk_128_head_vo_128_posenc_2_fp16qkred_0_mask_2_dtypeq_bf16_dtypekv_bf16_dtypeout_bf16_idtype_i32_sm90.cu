#include "hip/hip_runtime.h"
 // batch_paged_prefill_sm90 template inst
#include <flashinfer/attention/hopper/default_params.cuh>
#include <flashinfer/attention/hopper/prefill_sm90.cuh>
#include <flashinfer/attention/hopper/variants.cuh>
#include <flashinfer/cutlass_utils.cuh>


namespace flashinfer {

using DTypeQ = cutlass_dtype_t<hip_bfloat16>;
using DTypeKV = cutlass_dtype_t<hip_bfloat16>;
using DTypeO = cutlass_dtype_t<hip_bfloat16>;

using Params = BatchPrefillPagedParams<DTypeQ, DTypeKV, DTypeO, int32_t>;


template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kCustom,
     /*USE_SLIDING_WINDOW=*/true,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/true,
     LogitsSoftCap,
     Params>
    (Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kCustom,
     /*USE_SLIDING_WINDOW=*/true,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/false,
     LogitsSoftCap,
     Params>
    (Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kCustom,
     /*USE_SLIDING_WINDOW=*/false,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/true,
     LogitsSoftCap,
     Params>
    (Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kCustom,
     /*USE_SLIDING_WINDOW=*/false,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/false,
     LogitsSoftCap,
     Params>
    (Params& params, hipStream_t stream);
    


template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kCustom,
     /*USE_SLIDING_WINDOW=*/true,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/true,
     StandardAttention,
     Params>
    (Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kCustom,
     /*USE_SLIDING_WINDOW=*/true,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/false,
     StandardAttention,
     Params>
    (Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kCustom,
     /*USE_SLIDING_WINDOW=*/false,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/true,
     StandardAttention,
     Params>
    (Params& params, hipStream_t stream);

template hipError_t BatchPrefillWithPagedKVCacheDispatched
    <128,
     128,
     MaskMode::kCustom,
     /*USE_SLIDING_WINDOW=*/false,
     /*SAME_SCHEDULE_FOR_ALL_HEADS=*/false,
     StandardAttention,
     Params>
    (Params& params, hipStream_t stream);
    

}